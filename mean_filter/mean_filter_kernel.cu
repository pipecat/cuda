#include "hip/hip_runtime.h"
#include <c10/cuda/CUDAException.h>
#include <c10/cuda/CUDAStream.h>
#include <cassert>
#include <cstdio>


__global__
void mean_filter_kernel(unsigned char* output, unsigned char* input, int width, int height, int radius) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int channel = threadIdx.z;

    int baseOffset = channel * height * width;
    if (col < width && row < height) {

        int pixVal = 0;
        int pixels = 0;

        for (int blurRow=-radius; blurRow <= radius; blurRow += 1) {
            for (int blurCol=-radius; blurCol <= radius; blurCol += 1) {
                int curRow = row + blurRow;
                int curCol = col + blurCol;
                if (curRow >= 0 && curRow < height && curCol >=0 && curCol < width) {
                    pixVal += input[baseOffset + curRow * width + curCol];
                    pixels += 1;
                }
            }
        }

        output[baseOffset + row * width + col] = (unsigned char)(pixVal / pixels);
    }
}

__global__
void shift_filter_kernel(unsigned char* output, unsigned char* input, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int channel = threadIdx.z;

    int baseOffset = channel * height * width;

    int col_shift = (col - 128 >= 0) ? col - 128 : width - (128 - col);
    int row_shift = (row - 128 >= 0) ? row - 128 : height - (128 - row);

    // printf("%d %d %d %d", col, col_shift, row, row_shift);

    if (col < width && row < height) {
        output[baseOffset + row * width + col] = input[baseOffset + row_shift * width + col_shift];
    }
}


// helper function for ceiling unsigned integer division
inline unsigned int cdiv(unsigned int a, unsigned int b) {
  return (a + b - 1) / b;
}


torch::Tensor mean_filter(torch::Tensor image, int radius) {
    assert(image.device().type() == torch::kCUDA);
    assert(image.dtype() == torch::kByte);
    assert(radius > 0);

    const auto channels = image.size(0);
    const auto height = image.size(1);
    const auto width = image.size(2);

    auto result = torch::empty_like(image);

    dim3 threads_per_block(16, 16, channels);
    dim3 number_of_blocks(
        cdiv(width, threads_per_block.x),
        cdiv(height, threads_per_block.y)
    );

    mean_filter_kernel<<<number_of_blocks, threads_per_block, 0, torch::cuda::getCurrentCUDAStream()>>>(
        result.data_ptr<unsigned char>(),
        image.data_ptr<unsigned char>(),
        width,
        height,
        radius
    );

    // check CUDA error status (calls hipGetLastError())
    C10_CUDA_KERNEL_LAUNCH_CHECK();

    return result;
}

torch::Tensor shift_filter(torch::Tensor image) {
    assert(image.device().type() == torch::kCUDA);
    assert(image.dtype() == torch::kByte);
    
    const auto channels = image.size(0);
    const auto height = image.size(1);
    const auto width = image.size(2);

    auto result = torch::empty_like(image);

    dim3 thread_per_block(16, 16, channels);
    dim3 number_of_blocks(
        cdiv(width, thread_per_block.x),
        cdiv(height, thread_per_block.y)
    );

    shift_filter_kernel<<<number_of_blocks, thread_per_block, 0, torch::cuda::getCurrentCUDAStream()>>>(
        result.data_ptr<unsigned char>(),
        image.data_ptr<unsigned char>(),
        width,
        height
    );

    // check CUDA error status (calls hipGetLastError())
    C10_CUDA_KERNEL_LAUNCH_CHECK();

    return result;
}